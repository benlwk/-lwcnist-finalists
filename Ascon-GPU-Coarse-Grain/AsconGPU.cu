#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h> 
#include <time.h>  
#include <sys/time.h>
#include "params.h"
#include "permutations.h"
// #include "printstate.cuh"
#include "word.h"


__global__ void crypto_hash_gpu_ref(uint8_t* out, uint8_t* in ,
                uint32_t len) {
  /* initialize */
  uint32_t tid = threadIdx.x, bid = blockIdx.x; 
  uint32_t idx_in = bid*BLOCK_SIZE*MLEN + tid*MLEN;
  uint32_t idx_out = bid*BLOCK_SIZE*CRYPTO_BYTES + tid*CRYPTO_BYTES;
  state_t s;
  s.x0 = ASCON_HASH_IV;
  s.x1 = 0;
  s.x2 = 0;
  s.x3 = 0;
  s.x4 = 0;
  P12(&s);

  /* absorb full plaintext blocks */
  while (len >= ASCON_HASH_RATE) {
    s.x0 ^= LOADBYTES(in + idx_in , 8);
    P12(&s);
    in += ASCON_HASH_RATE;
    len -= ASCON_HASH_RATE;
  }
  /* absorb final plaintext block */
  s.x0 ^= LOADBYTES(in + idx_in , len);
  s.x0 ^= PAD(len);
  P12(&s);

  /* squeeze full output blocks */
  len = CRYPTO_BYTES;
  while (len > ASCON_HASH_RATE) {
    STOREBYTES(out + idx_out, s.x0, 8);
    P12(&s);
    out += ASCON_HASH_RATE;
    len -= ASCON_HASH_RATE;
  }
  /* squeeze final output block */
  STOREBYTES(out + idx_out, s.x0, len);
  // printstate("squeeze output", &s);
}


void init_buffer(uint8_t *buffer, uint32_t numbytes)
{
  int i, k;
  for (uint32_t i = 0; i < MAX_MESSAGE_LENGTH; i++) buffer[i] = 0;
  for(k=0; k<BATCH; k++) for (i = 0; i < numbytes; i++)
    // buffer[k*j*numbytes + j*numbytes + i] = rand()%256;
    buffer[k*numbytes + i] = (uint8_t)k + i;
    // buffer[k*numbytes + i] = (uint8_t)i;
}


int main(int argc, char* argv[]) {   
  uint8_t *h_msg, *h_digest, *d_msg, *d_digest, *h_state, *d_state;
  int i, k, mlen = MLEN, blocks, threads;
  hipEvent_t start, stop;
  float elapsed;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipMalloc((void**)&d_digest, BATCH*CRYPTO_BYTES*sizeof(uint8_t));
  hipMalloc((void**)&d_state, (uint64_t) BATCH * STATE_SIZE * sizeof(uint8_t));
  hipMalloc((void**)&d_msg, (uint64_t) BATCH * MAX_MESSAGE_LENGTH * sizeof(uint8_t)); 
   
  hipHostMalloc((void**)&h_digest, BATCH * CRYPTO_BYTES * sizeof(uint8_t));
  hipHostMalloc((void**)&h_msg, (uint64_t) BATCH * MAX_MESSAGE_LENGTH * sizeof(uint8_t));
  hipHostMalloc((void**)&h_state, (uint64_t) BATCH * STATE_SIZE * sizeof(uint8_t));
    
//    // Configure cache
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(crypto_hash_gpu), hipFuncCachePreferShared);
   
   threads = BLOCK_SIZE; 
   blocks = BATCH / BLOCK_SIZE;
   if(blocks == 0) blocks = 1;  // wklee, at least one block.
  printf("GPU photon-beetle: using %u blocks and %u threads\n", blocks, threads);

//    /* initialize random seed: */   
//   // srand (time(NULL));  // comment out this to yield a static poly elements.
  init_buffer(h_msg, mlen);
  // for(k=0; k<2; k++) {printf("\nbatch %u: ", k); for (i = 0; i <mlen; i++) printf("%x ", h_msg[k*mlen + i]);  }

  hipMemcpy(d_msg, h_msg, (uint64_t)BATCH * MAX_MESSAGE_LENGTH * sizeof(uint8_t), hipMemcpyHostToDevice);
  printf("\n Timing ASCON...MLEN: %u\n", MLEN);    
  hipEventRecord(start);         
  crypto_hash_gpu_ref<<<blocks, threads>>>(d_digest, d_msg, mlen);
  hipEventRecord(stop);
  hipEventSynchronize(stop);   
  hipEventElapsedTime(&elapsed, start, stop); 
  hipMemcpy(h_digest, d_digest, BATCH * CRYPTO_BYTES * sizeof(uint8_t), hipMemcpyDeviceToHost);    
#ifdef PERF
  printf("Latency (ms)\t Average (us) \t Hash/s \t TP (Gbps)\n" );
  printf("%.4f \t %.4f \t %.0f \t %.0f\n", elapsed, elapsed*1000/(BATCH), BATCH/elapsed, (double) 8*BATCH*MLEN/1024/1024/elapsed);     
#endif    
  
#ifdef DEBUG
  for(k=0; k<4; k++) {printf("\n batch %u\n", k);for (i = 0; i <CRYPTO_BYTES; i++) {printf("%x", h_digest[k*CRYPTO_BYTES + i]);}}
#endif    

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipDeviceReset();
  return 0;
}

